#include "hip/hip_runtime.h"
#include "common.cuh"

rtDeclareVariable(float3, bgColor, , );
rtDeclareVariable(RayData, rayData, rtPayload, );

RT_PROGRAM void miss() {
	rayData.result = bgColor;
}
