#include "hip/hip_runtime.h"
#include "common.cuh"

rtDeclareVariable(float4, background, , );

rtDeclareVariable(RayColorData, rayColorData, rtPayload, );

RT_PROGRAM void miss() {
	rayColorData.result = background;
}
