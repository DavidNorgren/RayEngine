#include "hip/hip_runtime.h"
#include "common.cuh"

rtTextureSampler<float4, 2> sky;

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(RayColorData, curColorData, rtPayload, );

RT_PROGRAM void miss() {

	float3 nDir = normalize(ray.direction);
	float theta = atan2f(nDir.x, nDir.z);
	float phi = M_PIf * 0.5f - acosf(nDir.y);
	float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v = 0.5f * (1.0f + sin(phi));
	curColorData.result = tex2D(sky, u, v);

}
