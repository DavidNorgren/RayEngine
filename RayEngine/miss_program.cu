#include "hip/hip_runtime.h"
#include "common.cuh"

rtDeclareVariable(float4, background, , );

rtDeclareVariable(RayColorData, curColorData, rtPayload, );

RT_PROGRAM void miss() {
	curColorData.result = background;
}
