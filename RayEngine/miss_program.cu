#include "hip/hip_runtime.h"
#include "common.cuh"

rtDeclareVariable(float4, backgroundColor, , );

rtDeclareVariable(RayData, rayData, rtPayload, );

RT_PROGRAM void miss() {
	rayData.result = backgroundColor;
}
