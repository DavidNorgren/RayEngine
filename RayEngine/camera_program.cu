#include "hip/hip_runtime.h"
#include "common.cuh"

rtBuffer<float4, 2> renderBuffer;
rtDeclareVariable(float, offset, , );
rtDeclareVariable(float3, org, , );
rtDeclareVariable(float3, xaxis, , );
rtDeclareVariable(float3, yaxis, , );
rtDeclareVariable(float3, zaxis, , );
rtDeclareVariable(rtObject, sceneObj, , );

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );

RT_PROGRAM void camera() {

	if (launchIndex.x < offset) {
		renderBuffer[launchIndex] = make_float4(0.f);
		return;
	}

	float2 d = (make_float2(launchIndex) / make_float2(launchDim)) * 2.f - 1.f;
	float3 rayOrg = org;
	float3 rayDir = d.x * xaxis + d.y * yaxis + zaxis;

	Ray ray = make_Ray(rayOrg, rayDir, 0, 0.1f, RT_DEFAULT_MAX);

	RayColorData data;
	data.reflectDepth = 0;
	data.refractDepth = 0;
	rtTrace(sceneObj, ray, data);

	renderBuffer[launchIndex] = data.result;

}