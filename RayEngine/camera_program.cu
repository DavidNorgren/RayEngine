#include "hip/hip_runtime.h"
#include "common.cuh"

rtDeclareVariable(float, partition, , );
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, xaxis, , );
rtDeclareVariable(float3, yaxis, , );
rtDeclareVariable(float3, zaxis, , );
rtBuffer<float4, 2> outputBuffer;
rtDeclareVariable(rtObject, sceneObj, , );

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );

RT_PROGRAM void camera() {
	if (launchIndex.x < launchDim.x * partition) {
		outputBuffer[launchIndex] = make_float4(0.f);
		return;
	}

	float2 d = (make_float2(launchIndex) / make_float2(launchDim)) * 2.f - 1.f;
	float3 rayOrg = eye;
	float3 rayDir = d.x * xaxis + d.y * yaxis + zaxis;

	optix::Ray ray = make_Ray(rayOrg, rayDir, 0, 0.01f, RT_DEFAULT_MAX);

	RayData rd;
	rtTrace(sceneObj, ray, rd);

	outputBuffer[launchIndex] = make_float4(rd.result, 1.f);
}