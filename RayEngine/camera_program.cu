#include "hip/hip_runtime.h"
#include "common.cuh"

rtBuffer<float4, 2> renderBuffer;
rtDeclareVariable(float, offset, , );
rtDeclareVariable(float, windowWidth, , );
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, xaxis, , );
rtDeclareVariable(float3, yaxis, , );
rtDeclareVariable(float3, zaxis, , );
rtDeclareVariable(rtObject, sceneObj, , );

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );

RT_PROGRAM void camera() {

	float2 d = (make_float2(offset + launchIndex.x, launchIndex.y) / make_float2(windowWidth, launchDim.y)) * 2.f - 1.f;
	float3 rayOrg = eye;
	float3 rayDir = d.x * xaxis + d.y * yaxis + zaxis;

	Ray ray = make_Ray(rayOrg, rayDir, 0, 0.01f, RT_DEFAULT_MAX);

	RayColorData data;
	data.depth = 0;
	rtTrace(sceneObj, ray, data);

	renderBuffer[launchIndex] = data.result;
}