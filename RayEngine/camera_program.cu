#include "hip/hip_runtime.h"
#include "common.cuh"

rtDeclareVariable(float, offset, , );
rtDeclareVariable(float, windowWidth, , );
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, xaxis, , );
rtDeclareVariable(float3, yaxis, , );
rtDeclareVariable(float3, zaxis, , );
rtBuffer<float4, 2> outputBuffer;
rtDeclareVariable(rtObject, sceneObj, , );

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );

RT_PROGRAM void camera() {

	float2 d = (make_float2(offset + launchIndex.x, launchIndex.y) / make_float2(windowWidth, launchDim.y)) * 2.f - 1.f;
	float3 rayOrg = eye;
	float3 rayDir = d.x * xaxis + d.y * yaxis + zaxis;

	optix::Ray ray = make_Ray(rayOrg, rayDir, 0, 0.01f, RT_DEFAULT_MAX);

	RayData rd;
	rtTrace(sceneObj, ray, rd);

	outputBuffer[launchIndex] = make_float4(rd.result, 1.f);
}