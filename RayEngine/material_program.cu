#include "hip/hip_runtime.h"
#include "common.cuh"

rtBuffer<Light> lights;
rtTextureSampler<float4, 2> sampler;
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float4, ambient, , );
rtDeclareVariable(float4, diffuse, , );
rtDeclareVariable(float, shininess, , );

rtDeclareVariable(float3, normal, attribute normal, );
rtDeclareVariable(float2, texCoord, attribute texCoord, );

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(RayData, rayData, rtPayload, );

RT_PROGRAM void closestHit() {

	// Set hit properties
	float3 hitPos = ray.origin + ray.tmax * ray.direction;
	float3 toEye = normalize(eye - hitPos);

	// Calculate color
	float4 totalDiffuse, totalSpecular;
	totalDiffuse = totalSpecular = make_float4(0.f);

	// Go through the lights
	for (int i = 0; i < lights.size(); i++) {
		Light light = lights[i];
		float3 incidence = normalize(light.position - hitPos);

		// Calculate attenuation (falloff)
		float distance = length(light.position - hitPos);
		float attenuation = max(1.f - distance / light.range, 0.f);

		// Diffuse factor
		float diffuse = max(dot(normal, incidence), 0.f) * attenuation;
		totalDiffuse += diffuse * light.color;

		// Specular factor
		if (shininess > 0.0) {
			float3 reflection = 2.f * dot(incidence, normal) * normal - incidence;
			float specular = pow(max(dot(reflection, toEye), 0.f), 1.f / shininess) * attenuation;
			totalSpecular += specular * light.color;
		}
	}

	// Create color
	float4 texColor = diffuse * tex2D(sampler, texCoord.x, texCoord.y);
	rayData.result = texColor * (ambient + totalDiffuse) + totalSpecular;
	rayData.result.w = texColor.w;

}
