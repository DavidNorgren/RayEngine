#include "hip/hip_runtime.h"
#include "common.cuh"

rtBuffer<Light> lights;
rtTextureSampler<float4, 2> sampler;
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float4, ambient, , );
rtDeclareVariable(float4, diffuse, , );
rtDeclareVariable(float, shininess, , );
rtDeclareVariable(rtObject, sceneObj, , );

rtDeclareVariable(float3, normal, attribute normal, );
rtDeclareVariable(float2, texCoord, attribute texCoord, );

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(RayColorData, rayColorData, rtPayload, );
rtDeclareVariable(RayShadowData, rayShadowData, rtPayload, );

RT_PROGRAM void anyHit() {

	rayShadowData.attenuation = 0.f;
	rtTerminateRay();

}

RT_PROGRAM void closestHit() {

	// Set hit properties
	float3 hitPos = ray.origin + ray.tmax * ray.direction;
	float3 toEye = normalize(eye - hitPos);

	// Calculate color
	float4 totalDiffuse, totalSpecular;
	totalDiffuse = totalSpecular = make_float4(0.f);

	// Go through the lights
	for (int i = 0; i < lights.size(); i++) {
		Light light = lights[i];
		float3 incidence = normalize(light.position - hitPos);

		// Calculate attenuation (falloff)
		float distance = length(light.position - hitPos);
		float attenuation = max(1.f - distance / light.range, 0.f);

		if (attenuation > 0.0) {

			// Cast shadow ray
			RayShadowData shadowData = { 1.f };
			optix::Ray shadowRay(hitPos, incidence, 1, 0.1f, distance);
			rtTrace(sceneObj, shadowRay, shadowData);

			if (shadowData.attenuation > 0.f) {

				attenuation *= shadowData.attenuation;

				// Diffuse factor
				float diffuse = max(dot(normal, incidence), 0.f) * attenuation;
				totalDiffuse += diffuse * light.color;

				// Specular factor
				if (shininess > 0.0) {
					float3 reflection = 2.f * dot(incidence, normal) * normal - incidence;
					float specular = pow(max(dot(reflection, toEye), 0.f), 1.f / shininess) * attenuation;
					totalSpecular += specular * light.color;
				}

			}

		}
	}

	// Create color
	float4 texColor = diffuse * tex2D(sampler, texCoord.x, texCoord.y);
	rayColorData.result = texColor * (ambient + totalDiffuse) + totalSpecular;
	rayColorData.result.w = texColor.w;

}
