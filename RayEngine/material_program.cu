#include "hip/hip_runtime.h"
#include "common.cuh"

rtBuffer<Light> lights;
rtTextureSampler<float4, 2> sampler;
rtDeclareVariable(float3, org, , );
rtDeclareVariable(float4, ambient, , );
rtDeclareVariable(float4, specular, , );
rtDeclareVariable(float4, diffuse, , );
rtDeclareVariable(float, shineExponent, , );
rtDeclareVariable(float, reflectIntensity, , );
rtDeclareVariable(float, refractIndex, , );
rtDeclareVariable(float4, sceneAmbient, , );
rtDeclareVariable(rtObject, sceneObj, , );
rtDeclareVariable(int, maxReflections, , );

rtDeclareVariable(float3, normal, attribute normal, );
rtDeclareVariable(float2, texCoord, attribute texCoord, );

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(RayColorData, curColorData, rtPayload, );
rtDeclareVariable(RayShadowData, curShadowData, rtPayload, );

RT_PROGRAM void anyHit() {

	float transColor = diffuse.w * tex2D(sampler, texCoord.x, texCoord.y).w;
	curShadowData.attenuation *= 1.f - transColor;

	if (curShadowData.attenuation == 0.f)
		rtTerminateRay();

}

RT_PROGRAM void closestHit() {

	// Set hit properties
	float3 hitPos = ray.origin + ray.tmax * ray.direction;
	float3 toEye = normalize(org - hitPos);

	// Calculate color
	float4 totalDiffuse, totalSpecular, totalReflect, totalRefract;
	totalDiffuse = totalSpecular = totalReflect = totalRefract = make_float4(0.f);

	// Go through the lights
	for (int i = 0; i < lights.size(); i++) {
		Light light = lights[i];
		float3 incidence = normalize(light.position - hitPos);

		// Calculate attenuation (falloff)
		float distance = length(light.position - hitPos);
		float attenuation = max(1.f - distance / light.range, 0.f);

		if (attenuation > 0.0) {

			// Cast shadow ray
			RayShadowData shadowData = { 1.f };
			Ray shadowRay(hitPos, incidence, 1, 0.01f, distance);
			rtTrace(sceneObj, shadowRay, shadowData);

			if (shadowData.attenuation > 0.f) {

				attenuation *= shadowData.attenuation;

				// Diffuse factor
				float diffuseFactor = max(dot(normal, incidence), 0.f) * attenuation;
				totalDiffuse += diffuseFactor * light.color;

				// Specular factor
				if (shineExponent > 0.0) {
					float3 reflection = -reflect(incidence, normal);
					float specularFactor = pow(max(dot(reflection, toEye), 0.f), shineExponent) * attenuation;
					totalSpecular += specularFactor * specular;
				}

			}

		}
	}

	// Reflection
	if (reflectIntensity > 0.f && curColorData.depth < maxReflections) {
		RayColorData reflectData;
		reflectData.depth = curColorData.depth + 1;
		Ray reflectRay(hitPos, reflect(ray.direction, normal), 0, 0.01f);
		rtTrace(sceneObj, reflectRay, reflectData);
		totalReflect = reflectData.result * reflectIntensity; // TODO: Use material reflectiveness
	}

	// Refract
	float transparency = diffuse.w * tex2D(sampler, texCoord.x, texCoord.y).w;
	if (transparency < 1.f) {

		float3 refractVector;
		if (!refract(refractVector, ray.direction, normal, refractIndex))
			refractVector = ray.direction;

		RayColorData refractData;
		refractData.depth = curColorData.depth;
		Ray refractRay(hitPos, refractVector, 0, 0.01f);
		rtTrace(sceneObj, refractRay, refractData);
		totalRefract = refractData.result * (1.f - transparency);

	}

	// Create color
	float4 texColor = diffuse * tex2D(sampler, texCoord.x, texCoord.y);
	curColorData.result = texColor * (sceneAmbient + ambient + totalDiffuse) + totalSpecular + totalReflect + totalRefract;


}
