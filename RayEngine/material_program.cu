#include "hip/hip_runtime.h"
#include "common.cuh"

rtDeclareVariable(float3, normal, attribute normal, );
rtDeclareVariable(RayData, rayData, rtPayload, );

RT_PROGRAM void closestHit() {
	rayData.result = normal * 0.5f + 0.5f;
}
