#include "hip/hip_runtime.h"
#include "common.cuh"

rtBuffer<Light> lights;
rtTextureSampler<float4, 2> sampler;
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float4, ambient, , );
rtDeclareVariable(float4, specular, , );
rtDeclareVariable(float4, diffuse, , );
rtDeclareVariable(float, shininess, , );
rtDeclareVariable(float4, sceneAmbient, , );
rtDeclareVariable(rtObject, sceneObj, , );

rtDeclareVariable(float3, normal, attribute normal, );
rtDeclareVariable(float2, texCoord, attribute texCoord, );

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(RayColorData, curColorData, rtPayload, );
rtDeclareVariable(RayShadowData, curShadowData, rtPayload, );

RT_PROGRAM void anyHit() {

	curShadowData.attenuation = 0.f;
	rtTerminateRay();

}

RT_PROGRAM void closestHit() {

	// Set hit properties
	float3 hitPos = ray.origin + ray.tmax * ray.direction;
	float3 toEye = normalize(eye - hitPos);

	// Calculate color
	float4 totalDiffuse, totalSpecular, totalReflect;
	totalDiffuse = totalSpecular = totalReflect = make_float4(0.f);

	// Go through the lights
	for (int i = 0; i < lights.size(); i++) {
		Light light = lights[i];
		float3 incidence = normalize(light.position - hitPos);

		// Calculate attenuation (falloff)
		float distance = length(light.position - hitPos);
		float attenuation = max(1.f - distance / light.range, 0.f);

		if (attenuation > 0.0) {

			// Cast shadow ray
			RayShadowData shadowData = { 1.f };
			Ray shadowRay(hitPos, incidence, 1, 0.01f, distance);
			rtTrace(sceneObj, shadowRay, shadowData);

			if (shadowData.attenuation > 0.f) {

				attenuation *= shadowData.attenuation;

				// Diffuse factor
				float diffuseFactor = max(dot(normal, incidence), 0.f) * attenuation;
				totalDiffuse += diffuseFactor * light.color;

				// Specular factor
				if (shininess > 0.0) {
					float3 reflection = 2.f * dot(incidence, normal) * normal - incidence;
					float specularFactor = pow(max(dot(reflection, toEye), 0.f), shininess) * attenuation;
					totalSpecular += specularFactor * specular;
				}

			}

		}
	}

	// Reflection
	if (curColorData.depth < 1) {
		RayColorData reflectData;
		reflectData.depth = curColorData.depth + 1;
		Ray reflectRay(hitPos, reflect(ray.direction, normal), 0, 0.01f);
		rtTrace(sceneObj, reflectRay, reflectData);
		totalReflect = reflectData.result * make_float4(0.1f);
	}

	// Create color
	float4 texColor = diffuse * tex2D(sampler, texCoord.x, texCoord.y);
	curColorData.result = texColor * (sceneAmbient + ambient + totalDiffuse) + totalSpecular + totalReflect;
	curColorData.result.w = texColor.w;

}
