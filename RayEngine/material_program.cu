#include "hip/hip_runtime.h"
#include "common.cuh"

rtTextureSampler<float4, 2> sampler;
rtDeclareVariable(float3, normal, attribute normal, );
rtDeclareVariable(float2, texCoord, attribute texCoord, );
rtDeclareVariable(RayData, rayData, rtPayload, );

RT_PROGRAM void closestHit() {
	rayData.result = make_float3(tex2D(sampler, texCoord.x, texCoord.y));
}
