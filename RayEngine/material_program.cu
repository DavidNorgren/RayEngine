#include "hip/hip_runtime.h"
#include "common.cuh"
#include "random.cuh"

rtBuffer<Light> lights;
rtTextureSampler<float4, 2> sampler;
rtDeclareVariable(float3, org, , );
rtDeclareVariable(float4, ambient, , );
rtDeclareVariable(float4, specular, , );
rtDeclareVariable(float4, diffuse, , );
rtDeclareVariable(float, shineExponent, , );
rtDeclareVariable(float, reflectIntensity, , );
rtDeclareVariable(float, refractIndex, , );
rtDeclareVariable(float4, sceneAmbient, , );
rtDeclareVariable(rtObject, sceneObj, , );
rtDeclareVariable(int, maxReflections, , );
rtDeclareVariable(int, maxRefractions, , );
rtDeclareVariable(float, aoRadius, , );
rtTextureSampler<float4, 2> aoNoise;

rtDeclareVariable(float3, normal, attribute normal, );
rtDeclareVariable(float2, texCoord, attribute texCoord, );

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(RayColorData, curColorData, rtPayload, );
rtDeclareVariable(RayShadowData, curShadowData, rtPayload, );

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );

RT_PROGRAM void anyHit() {

	curShadowData.attenuation *= 1.f - diffuse.w * tex2D(sampler, texCoord.x, texCoord.y).w;

	if (curShadowData.attenuation == 0.f)
		rtTerminateRay();

}

RT_PROGRAM void closestHit() {

	// Set hit properties
	float3 hitPos = ray.origin + ray.tmax * ray.direction;
	float3 toEye = normalize(org - hitPos);

	// Calculate color
	float4 texture = diffuse * tex2D(sampler, texCoord.x, texCoord.y);
	float transparency = 1.f - texture.w;
	float4 totalDiffuse, totalSpecular, totalReflect, totalRefract;
	totalDiffuse = totalSpecular = totalReflect = totalRefract = make_float4(0.f);

	//// Light contribution ////

	for (int i = 0; i < lights.size(); i++) {

		Light light = lights[i];
		float3 incidence = normalize(light.position - hitPos);

		// Calculate attenuation (falloff)
		float distance = length(light.position - hitPos);
		float attenuation = max(1.f - distance / light.range, 0.f);

		if (attenuation > 0.0) {

			// Cast ray to find blocking objects
			RayShadowData shadowData;
			shadowData.attenuation = attenuation;
			Ray shadowRay(hitPos, incidence, 1, 0.1f, distance);
			rtTrace(sceneObj, shadowRay, shadowData);

			// The ray was not fully absorbed, add light contribution
			if (shadowData.attenuation > 0.f) {

				// Diffuse factor
				float diffuseFactor = max(dot(normal, incidence), 0.f) * shadowData.attenuation;
				totalDiffuse += diffuseFactor * light.color;

				// Specular factor
				if (shineExponent > 0.f) {
					float3 reflection = -reflect(incidence, normal);
					float specularFactor = pow(max(dot(reflection, toEye), 0.f), shineExponent) * shadowData.attenuation;
					totalSpecular += specularFactor * specular;
				}

			}

		}

	}

	//// Reflection ////

	if (reflectIntensity > 0.f && curColorData.reflectDepth < maxReflections) {

		float3 reflectVector = reflect(ray.direction, normal);

		RayColorData reflectData;
		reflectData.reflectDepth = curColorData.reflectDepth + 1;
		reflectData.refractDepth = curColorData.refractDepth;

		Ray reflectRay(hitPos, reflectVector, 0, 0.1f);
		rtTrace(sceneObj, reflectRay, reflectData);
		totalReflect = reflectData.result * reflectIntensity;

	}

	//// Refraction ////

	if (transparency > 0.f && curColorData.refractDepth < maxRefractions) {

		float3 refractVector;
		if (!refract(refractVector, ray.direction, normal, refractIndex))
			refractVector = ray.direction;

		RayColorData refractData;
		refractData.reflectDepth = curColorData.reflectDepth;
		refractData.refractDepth = curColorData.refractDepth + 1;

		Ray refractRay(hitPos, refractVector, 0, 0.1f);
		rtTrace(sceneObj, refractRay, refractData);
		totalRefract = refractData.result * transparency;

	}

	/// Ambient occlusion ////

	float2 pos = (make_float2(launchIndex) / make_float2(launchDim)) * 20.f;
	float4 noise = tex2D(aoNoise, pos.x, pos.y);
	float occluded = 0.f;
	Onb onb(normal);

	int samples = 16;
	float invSamples = 1.f / samples;
	for (int i = 0; i < samples; i++) {

		float u1 = (float(i % 4) + noise.x) * (1.f / 5.f);
		float u2 = (float(i / 4) + noise.y) * (1.f / 5.f);
		float3 sampleVector;
		cosine_sample_hemisphere(u1, u2, sampleVector);
		onb.inverse_transform(sampleVector);

		RayShadowData sampleData;
		sampleData.attenuation = 1.f;
		Ray sampleRay(hitPos, sampleVector, 1, 0.01f, aoRadius);
		rtTrace(sceneObj, sampleRay, sampleData);

		occluded += 1.f - sampleData.attenuation;

	}

	float aoPower = 2.f;
	occluded = 1.f - pow(1.f - occluded * invSamples, aoPower);

	// Create color
	curColorData.result = texture * (sceneAmbient + ambient + totalDiffuse) * (1.f - occluded) * (1.f - transparency) + totalSpecular + totalReflect + totalRefract;
	curColorData.result.w = 1.f;

}
