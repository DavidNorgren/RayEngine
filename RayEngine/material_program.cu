#include "hip/hip_runtime.h"
#include "common.cuh"

rtBuffer<Light> lights;
rtTextureSampler<float4, 2> sampler;
rtDeclareVariable(float3, org, , );
rtDeclareVariable(float4, ambient, , );
rtDeclareVariable(float4, specular, , );
rtDeclareVariable(float4, diffuse, , );
rtDeclareVariable(float, shineExponent, , );
rtDeclareVariable(float, reflectIntensity, , );
rtDeclareVariable(float, refractIndex, , );
rtDeclareVariable(float4, sceneAmbient, , );
rtDeclareVariable(rtObject, sceneObj, , );
rtDeclareVariable(int, maxReflections, , );
rtDeclareVariable(int, maxRefractions, , );

rtDeclareVariable(float3, normal, attribute normal, );
rtDeclareVariable(float2, texCoord, attribute texCoord, );

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(RayColorData, curColorData, rtPayload, );
rtDeclareVariable(RayShadowData, curShadowData, rtPayload, );

RT_PROGRAM void anyHit() {

	curShadowData.attenuation *= 1.f - diffuse.w * tex2D(sampler, texCoord.x, texCoord.y).w;

	if (curShadowData.attenuation == 0.f)
		rtTerminateRay();

}

RT_PROGRAM void closestHit() {

	// Set hit properties
	float3 hitPos = ray.origin + ray.tmax * ray.direction;
	float3 toEye = normalize(org - hitPos);

	// Calculate color
	float4 texture = diffuse * tex2D(sampler, texCoord.x, texCoord.y);
	float transparency = 1.f - texture.w;
	float4 totalDiffuse, totalSpecular, totalReflect, totalRefract;
	totalDiffuse = totalSpecular = totalReflect = totalRefract = make_float4(0.f);

	//// Light contribution ////

	for (int i = 0; i < lights.size(); i++) {
		Light light = lights[i];
		float3 incidence = normalize(light.position - hitPos);

		// Calculate attenuation (falloff)
		float distance = length(light.position - hitPos);
		float attenuation = max(1.f - distance / light.range, 0.f);

		if (attenuation > 0.0) {

			// Cast ray to find blocking objects
			RayShadowData shadowData;
			shadowData.attenuation = 1.f;
			Ray shadowRay(hitPos, incidence, 1, 0.01f, distance);
			rtTrace(sceneObj, shadowRay, shadowData);

			// The ray was not fully absorbed, add light contribution
			if (shadowData.attenuation > 0.f) {

				attenuation *= shadowData.attenuation;

				// Diffuse factor
				float diffuseFactor = max(dot(normal, incidence), 0.f) * attenuation;
				totalDiffuse += diffuseFactor * light.color;

				// Specular factor
				if (shineExponent > 0.0) {
					float3 reflection = -reflect(incidence, normal);
					float specularFactor = pow(max(dot(reflection, toEye), 0.f), shineExponent) * attenuation;
					totalSpecular += specularFactor * specular;
				}

			}

		}
	}

	//// Reflection ////

	if (reflectIntensity > 0.f && curColorData.reflectDepth < maxReflections) {
		RayColorData reflectData;
		reflectData.reflectDepth = curColorData.reflectDepth + 1;
		Ray reflectRay(hitPos, reflect(ray.direction, normal), 0, 0.01f);
		rtTrace(sceneObj, reflectRay, reflectData);
		totalReflect = reflectData.result * reflectIntensity; // TODO: Use material reflectiveness
	}

	//// Refraction ////

	if (transparency > 0.f && curColorData.refractDepth < maxRefractions) {

		float3 refractVector;
		if (!refract(refractVector, ray.direction, normal, refractIndex))
			refractVector = ray.direction;

		RayColorData refractData;
		refractData.refractDepth = curColorData.refractDepth+ 1;
		Ray refractRay(hitPos, refractVector, 0, 0.01f);
		rtTrace(sceneObj, refractRay, refractData);
		totalRefract = refractData.result * transparency;

	}

	// Create color
	curColorData.result = texture * (sceneAmbient + ambient + totalDiffuse) * (1.f - transparency) + totalSpecular + totalReflect + totalRefract;
	//curColorData.result.w = 1.f;

}
