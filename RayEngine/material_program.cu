#include "hip/hip_runtime.h"
#include "common.cuh"
#include "random.cuh"

rtDeclareVariable(float, offset, , );
rtDeclareVariable(float, windowWidth, , );
rtBuffer<Light> lights;
rtTextureSampler<float4, 2> sampler;
rtDeclareVariable(float3, org, , );
rtDeclareVariable(float4, ambient, , );
rtDeclareVariable(float4, specular, , );
rtDeclareVariable(float4, diffuse, , );
rtDeclareVariable(float, shineExponent, , );
rtDeclareVariable(float, reflectIntensity, , );
rtDeclareVariable(float, refractIndex, , );
rtDeclareVariable(float4, sceneAmbient, , );
rtDeclareVariable(rtObject, sceneObj, , );
rtDeclareVariable(int, enableReflections, , );
rtDeclareVariable(int, maxReflections, , );
rtDeclareVariable(int, enableRefractions, , );
rtDeclareVariable(int, maxRefractions, , );
rtDeclareVariable(int, enableAo, , );
rtDeclareVariable(float, aoRadius, , );
rtDeclareVariable(float, aoPower, , );
rtDeclareVariable(int, aoSamples, , );
rtDeclareVariable(int, aoSamplesSqrt, , );
rtTextureSampler<float4, 2> aoNoise;
rtDeclareVariable(float, aoNoiseScale, , );

rtDeclareVariable(float3, normal, attribute normal, );
rtDeclareVariable(float2, texCoord, attribute texCoord, );

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(RayColorData, curColorData, rtPayload, );
rtDeclareVariable(RayShadowData, curShadowData, rtPayload, );

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );

RT_PROGRAM void anyHit() {

	curShadowData.attenuation *= 1.f - diffuse.w * tex2D(sampler, texCoord.x, texCoord.y).w;

	if (curShadowData.attenuation == 0.f)
		rtTerminateRay();

}

RT_PROGRAM void closestHit() {

	// Set hit properties
	float3 hitPos = ray.origin + ray.tmax * ray.direction;
	float3 toEye = normalize(org - hitPos);

	// Calculate color
	float4 texture = diffuse * tex2D(sampler, texCoord.x, texCoord.y);
	float transparency = 1.f - texture.w;
	float4 totalDiffuse, totalSpecular, totalReflect, totalRefract;
	totalDiffuse = totalSpecular = totalReflect = totalRefract = make_float4(0.f);

	//// Light contribution ////

	for (int i = 0; i < lights.size(); i++) {

		Light light = lights[i];
		float3 incidence = normalize(light.position - hitPos);

		// Calculate attenuation (falloff)
		float distance = length(light.position - hitPos);
		float attenuation = max(1.f - distance / light.range, 0.f);

		if (attenuation > 0.0) {

			// Cast ray to find blocking objects
			RayShadowData shadowData;
			shadowData.attenuation = attenuation;
			Ray shadowRay(hitPos, incidence, 1, 0.01f, distance);
			rtTrace(sceneObj, shadowRay, shadowData);

			// The ray was not fully absorbed, add light contribution
			if (shadowData.attenuation > 0.f) {

				// Diffuse factor
				float diffuseFactor = max(dot(normal, incidence), 0.f) * shadowData.attenuation;
				totalDiffuse += diffuseFactor * light.color;

				// Specular factor
				if (shineExponent > 0.f) {
					float3 reflection = -reflect(incidence, normal);
					float specularFactor = pow(max(dot(reflection, toEye), 0.f), shineExponent) * shadowData.attenuation;
					totalSpecular += specularFactor * specular;
				}

			}

		}

	}

	//// Reflection ////

	if (enableReflections && reflectIntensity > 0.f && curColorData.reflectDepth < maxReflections) {

		float3 reflectVector = reflect(ray.direction, normal);

		RayColorData reflectData;
		reflectData.reflectDepth = curColorData.reflectDepth + 1;
		reflectData.refractDepth = curColorData.refractDepth;

		Ray reflectRay(hitPos, reflectVector, 0, 0.01f);
		rtTrace(sceneObj, reflectRay, reflectData);
		totalReflect = reflectData.result * reflectIntensity;

	}

	//// Refraction ////

	if (enableRefractions && transparency > 0.f && curColorData.refractDepth < maxRefractions) {

		float3 refractVector;
		if (!refract(refractVector, ray.direction, normal, refractIndex))
			refractVector = ray.direction;

		RayColorData refractData;
		refractData.reflectDepth = curColorData.reflectDepth;
		refractData.refractDepth = curColorData.refractDepth + 1;

		Ray refractRay(hitPos, refractVector, 0, 0.01f);
		rtTrace(sceneObj, refractRay, refractData);
		totalRefract = refractData.result * transparency;

	}

	/// Ambient occlusion ////

	float occluded = 0.f;

	if (enableAo) {

		float invSamples = 1.f / aoSamples;
		float invSamplesSqrt = 1.f / aoSamplesSqrt;
		float2 noiseTexCoord = (make_float2(launchIndex) / make_float2(launchDim)) * aoNoiseScale;
		float4 noise = tex2D(aoNoise, noiseTexCoord.x, noiseTexCoord.y);
		Onb onb(normal);

		for (int i = 0; i < aoSamples; i++) {

			float u1 = (float(i % aoSamplesSqrt) + noise.x) * invSamplesSqrt;
			float u2 = (float(i / aoSamplesSqrt) + noise.y) * invSamplesSqrt;
			float3 sampleVector;
			cosine_sample_hemisphere(u1, u2, sampleVector);
			onb.inverse_transform(sampleVector);

			RayShadowData sampleData;
			sampleData.attenuation = 1.f;
			Ray sampleRay(hitPos, sampleVector, 1, 0.01f, aoRadius);
			rtTrace(sceneObj, sampleRay, sampleData);

			occluded += 1.f - sampleData.attenuation;

		}

		occluded *= invSamples * aoPower;

	}

	// Create color
	curColorData.result = texture * (sceneAmbient + ambient + totalDiffuse) * (1.f - occluded) * (1.f - transparency) + totalSpecular + totalReflect + totalRefract;
	curColorData.result.w = 1.f;

}
