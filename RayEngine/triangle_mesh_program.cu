#include "hip/hip_runtime.h"
#include "common.cuh"

rtBuffer<float3, 1> posData;
rtBuffer<float3, 1> normalData;
rtBuffer<float2, 1> texCoordData;
rtBuffer<uint3, 1> indexData;

rtDeclareVariable(float3, normal, attribute normal, );
rtDeclareVariable(float2, texCoord, attribute texCoord, );

rtDeclareVariable(Ray, ray, rtCurrentRay, );

RT_PROGRAM void intersect(int primId) {

	uint3 prim = indexData[primId];
	float3 p0 = posData[prim.x];
	float3 p1 = posData[prim.y];
	float3 p2 = posData[prim.z];

	float3 n;
	float t, u, v;

	if (intersect_triangle(ray, p0, p1, p2, n, t, u, v)) {

		if (rtPotentialIntersection(t)) {

			// Normal
			float3 n0 = normalData[prim.x];
			float3 n1 = normalData[prim.y];
			float3 n2 = normalData[prim.z];
			normal = normalize(rtTransformVector(RT_OBJECT_TO_WORLD, n0 * (1.f - u - v) + n1 * u + n2 * v));

			// Texture coordinate
			float2 t0 = texCoordData[prim.x];
			float2 t1 = texCoordData[prim.y];
			float2 t2 = texCoordData[prim.z];
			texCoord = t0 * (1.f - u - v) + t1 * u + t2 * v;

			rtReportIntersection(0);
		}
	}

}

RT_PROGRAM void bounds(int primId, float result[6]) {

	uint3 prim = indexData[primId];
	float3 p0 = posData[prim.x];
	float3 p1 = posData[prim.y];
	float3 p2 = posData[prim.z];

	optix::Aabb* aabb = (optix::Aabb*)result;
	aabb->m_min = fminf(fminf(p0, p1), p2);
	aabb->m_max = fmaxf(fmaxf(p0, p1), p2);
}